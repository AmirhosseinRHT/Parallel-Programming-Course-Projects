#include "hip/hip_runtime.h"
%%writefile  example.cu
#include <iostream>
#include <fstream>
#include <cmath>
#include <hiprand/hiprand_kernel.h>

//#define M_PI 3.14159265358979323846f

struct Vec3 {
    float x, y, z;

    __host__ __device__ Vec3() : x(0), y(0), z(0) {}
    __host__ __device__ Vec3(float x, float y, float z) : x(x), y(y), z(z) {}

    __host__ __device__ Vec3 operator+(const Vec3& v) const {
        return Vec3(x + v.x, y + v.y, z + v.z);
    }
    __host__ __device__ Vec3 operator-(const Vec3& v) const {
        return Vec3(x - v.x, y - v.y, z - v.z);
    }
    __host__ __device__ Vec3 operator*(float t) const {
        return Vec3(x * t, y * t, z * t);
    }
    __host__ __device__ Vec3 operator/(float t) const {
        return Vec3(x / t, y / t, z / t);
    }
    __host__ __device__ float dot(const Vec3& v) const {
        return x * v.x + y * v.y + z * v.z;
    }
    __host__ __device__ Vec3 cross(const Vec3& v) const {
        return Vec3(
                y * v.z - z * v.y,
                z * v.x - x * v.z,
                x * v.y - y * v.x
        );
    }
    __host__ __device__ float length() const {
        return sqrtf(x * x + y * y + z * z);
    }
    __host__ __device__ Vec3 normalize() const {
        float len = length();
        return *this / len;
    }
    __host__ __device__ Vec3 operator*(const Vec3& v) const {
        return Vec3(x * v.x, y * v.y, z * v.z);
    }

    __host__ __device__ friend Vec3 operator*(float t, const Vec3& v) {
        return Vec3(t * v.x, t * v.y, t * v.z);
    }
};

struct Ray {
    Vec3 origin;
    Vec3 direction;

    __host__ __device__ Ray() {}
    __host__ __device__ Ray(const Vec3& o, const Vec3& d) : origin(o), direction(d) {}

    __host__ __device__ Vec3 at(float t) const {
        return origin + direction * t;
    }
};

#define SPHERE 0
#define PLANE 1

struct Hittable {
    int type;        // Object type: SPHERE or PLANE
    Vec3 center;     // For sphere and plane (point on the plane)
    Vec3 normal;     // For plane normal (for PLANE type)
    float radius;    // For sphere
    Vec3 color;      // Material color
};

__device__ bool hitSphere(const Hittable& sphere, const Ray& r, float t_min, float t_max, float& t, Vec3& normal) {
    Vec3 oc = r.origin - sphere.center;
    float a = r.direction.dot(r.direction);
    float half_b = oc.dot(r.direction);
    float c = oc.dot(oc) - sphere.radius * sphere.radius;
    float discriminant = half_b * half_b - a * c;
    if (discriminant > 0) {
        float sqrt_d = sqrtf(discriminant);
        float root = (-half_b - sqrt_d) / a;
        if (root < t_max && root > t_min) {
            t = root;
            Vec3 hitPoint = r.at(t);
            normal = (hitPoint - sphere.center).normalize();
            return true;
        }
        root = (-half_b + sqrt_d) / a;
        if (root < t_max && root > t_min) {
            t = root;
            Vec3 hitPoint = r.at(t);
            normal = (hitPoint - sphere.center).normalize();
            return true;
        }
    }
    return false;
}

__device__ bool hitPlane(const Hittable& plane, const Ray& r, float t_min, float t_max, float& t, Vec3& normal) {
    float denom = plane.normal.dot(r.direction);
    if (fabsf(denom) > 1e-6f) { // Not parallel
        t = (plane.center - r.origin).dot(plane.normal) / denom;
        if (t < t_max && t > t_min) {
            normal = plane.normal;
            return true;
        }
    }
    return false;
}


__device__ bool does_shadow_intersect(Vec3 point,Vec3 light_pos, Hittable* objects, int num_objects){
    float t_min = 0.001f;
    float t_max = 1e20f;
    Vec3 shadow_dir =  light_pos - point;
    shadow_dir = shadow_dir.normalize();
    Ray shadow(point, shadow_dir);
    float t;
    Vec3 temp_normal;
    for(int i = 0; i < num_objects;i++){
        if(objects[i].type == SPHERE){
            if(hitSphere(objects[i],shadow, t_min, t_max, t, temp_normal)){
                return true;
            }

        }
    }
    return false;
}

__device__ Vec3 rayColor(const Ray& r, Hittable* objects, int num_objects, Vec3 light_pos) {
    float t_min = 0.001f;
    float t_max = 1e20f;
    float closest_t = t_max;
    Vec3 color(0, 0, 0);
    Vec3 normal;
    int hit_index = -1;

    // Find closest hit
    for (int i = 0; i < num_objects; ++i) {
        float t;
        Vec3 temp_normal;
        bool hit = false;

        if (objects[i].type == SPHERE) {
            hit = hitSphere(objects[i], r, t_min, closest_t, t, temp_normal);
        } else if (objects[i].type == PLANE) {
            hit = hitPlane(objects[i], r, t_min, closest_t, t, temp_normal);
        }

        if (hit) {
            closest_t = t;
            normal = temp_normal;
            color = objects[i].color;
            hit_index = i;
        }
    }

    if (hit_index >= 0) {

        Vec3 shadow_color(0,0,0);
        Vec3 hit_point = r.at(closest_t);
        Vec3 diffuse;
        if(does_shadow_intersect(hit_point, light_pos, objects,num_objects)){
            diffuse = shadow_color;
        }
        else{
            Vec3 light_dir = (light_pos - r.at(closest_t)).normalize();
            float intensity = fmaxf(0.0f, normal.dot(light_dir));
            diffuse = intensity * color;
        }
        
       

        Vec3 ambient = 0.1f * color;

        Vec3 result_color = ambient + diffuse;

        return result_color;
    }

    Vec3 unit_direction = r.direction.normalize();
    float t = 0.5f * (unit_direction.y + 1.0f);
    return (1.0f - t) * Vec3(1.0f, 1.0f, 1.0f) + t * Vec3(0.5f, 0.7f, 1.0f); // Sky gradient
}

__global__ void renderKernel(Vec3* pixels, int width, int height, Hittable* objects, int num_objects, Vec3 light_pos,Vec3 lookfrom, Vec3 lookat, float vfov, Vec3 up) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;

    int index = y * width + x;
    

    float aspect_ratio = float(width) / float(height);
    float viewport_height = 2.0f * tan(vfov / 2);
    float viewport_width = aspect_ratio * viewport_height;
    float focal_length = (lookfrom - lookat).length();
    Vec3 w = (lookfrom - lookat).normalize();
    Vec3 u = up.cross(w).normalize();
    Vec3 v = w.cross(u);
   
    Vec3 h = viewport_width * u;
    Vec3 vertical = viewport_height * v;

    Vec3 viewport_upper_left  = lookfrom - (w * focal_length) - (h / 2) - (vertical / 2);

    Vec3 d_u = h / float(width);
    Vec3 d_v = vertical / float(height);

    Vec3 pixel_location = viewport_upper_left  + (x + 0.5f) * d_u + (y + 0.5f) * d_v;

    
    Vec3 direction = (pixel_location - lookfrom).normalize();
    Ray r(lookfrom, direction);

    Vec3 color = rayColor(r, objects, num_objects, light_pos);

    color = Vec3(sqrtf(color.x), sqrtf(color.y), sqrtf(color.z));
    pixels[index] = color;
}

void saveToPPM(const Vec3* pixels, int width, int height, const std::string& filename) {
    std::ofstream outFile(filename, std::ios::out | std::ios::binary);
    outFile << "P6\n" << width << " " << height << "\n255\n";

    for (int j = height - 1; j >= 0; --j) { // Flip the image vertically
        for (int i = 0; i < width; ++i) {
            int index = j * width + i;
            unsigned char r = static_cast<unsigned char>(255.99f * fminf(fmaxf(pixels[index].x, 0.0f), 1.0f));
            unsigned char g = static_cast<unsigned char>(255.99f * fminf(fmaxf(pixels[index].y, 0.0f), 1.0f));
            unsigned char b = static_cast<unsigned char>(255.99f * fminf(fmaxf(pixels[index].z, 0.0f), 1.0f));
            outFile << r << g << b;
        }
    }
    outFile.close();
}

int main() {
    float vfov = 120 * (M_PI/180.0);
    Vec3 up(0,1,0);
    Vec3 look_from(0,0,1);
    Vec3 look_at(0,0,-1);


    const int width = 800;
    const int height = 600;
    size_t numPixels = width * height;
    Vec3* pixels;

    hipMallocManaged(&pixels, numPixels * sizeof(Vec3));

    const int num_objects = 4;
    Hittable* objects;
    hipMallocManaged(&objects, num_objects * sizeof(Hittable));

    objects[0].type = SPHERE;
    objects[0].center = Vec3(0.0f, 0.0f, -1.5f);
    objects[0].radius = 0.5f;
    objects[0].color = Vec3(0.8f, 0.1f, 0.1f); // Red

    objects[1].type = SPHERE;
    objects[1].center = Vec3(-1.0f, 0.0f, -2.0f);
    objects[1].radius = 0.5f;
    objects[1].color = Vec3(0.1f, 0.1f, 0.8f); // Blue

    objects[2].type = SPHERE;
    objects[2].center = Vec3(1.0f, 0.0f, -2.0f);
    objects[2].radius = 0.5f;
    objects[2].color = Vec3(0.1f, 0.8f, 0.1f); // Green

    objects[3].type = PLANE;
    objects[3].center = Vec3(0.0f, -0.5f, 0.0f); // Point on the plane
    objects[3].normal = Vec3(0.0f, 1.0f, 0.0f);  // Upward normal
    objects[3].color = Vec3(0.8f, 0.8f, 0.8f);   // Gray

    Vec3 light_pos = Vec3(5.0f, 5.0f, -5.0f);

    dim3 blockSize(16, 16);
    dim3 numBlocks((width + blockSize.x - 1) / blockSize.x,
                   (height + blockSize.y - 1) / blockSize.y);
    
    renderKernel<<<numBlocks, blockSize>>>(pixels, width, height, objects, num_objects, light_pos,look_from,look_at,vfov,up);
    hipDeviceSynchronize();

    saveToPPM(pixels, width, height, "output.ppm");

    hipFree(pixels);
    hipFree(objects);

    return 0;
}
